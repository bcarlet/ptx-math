#include "cuda_check.hpp"

#include <cstdlib>
#include <cstdio>

void cuda_err_check(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA error in file %s on line %d: %s\n",
                file, line, hipGetErrorString(code));

        exit(-1);
    }
}
