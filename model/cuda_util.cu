#include "cuda_util.hpp"

#include <cstdlib>
#include <cstdio>

void cuda_err_check(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA error in file %s on line %d: %s\n",
                file, line, hipGetErrorString(code));

        exit(-1);
    }
}
