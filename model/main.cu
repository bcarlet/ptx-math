#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

#include "cuda_check.hpp"

__global__
static void sine(int n, float *x)
{
    const int stride = blockDim.x * gridDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += stride)
    {
        asm("sin.approx.f32 %0, %0;" : "+f"(x[i]));
    }
}

int main()
{
    const int x_size = 1 << 20;
    float *x;

    CUDA_CHECK(hipMallocManaged(&x, x_size * sizeof(float)));

    const float step = 0.000001f;

    for (int i = 0; i < x_size; i++)
    {
        x[i] = i * step;
    }

    const int block_dim = 256;
    const int grid_dim = (x_size + block_dim - 1) / block_dim;

    sine<<<grid_dim, block_dim>>>(x_size, x);
    CUDA_CHECK(hipPeekAtLastError());

    CUDA_CHECK(hipDeviceSynchronize());

    float max_error = 0.0f;

    for (int i = 0; i < x_size; i++)
    {
        const float expected = sin(i * step);

        max_error = fmax(max_error, fabs(x[i] - expected));
    }

    printf("Max error: %.10f\n", max_error);

    CUDA_CHECK(hipFree(x));

    return 0;
}
