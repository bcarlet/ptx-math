#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

#include "cuda_util.hpp"
#include "ptx.hpp"

__global__
static void sine(int n, float *x)
{
    int i, stride;

    GRID_STRIDE_LOOP(i, stride, n)
    {
        sin_approx_f32(x + i);
    }
}

int main()
{
    const int x_size = 1 << 20;
    float *x;

    CUDA_CHECK(hipMallocManaged(&x, x_size * sizeof(float)));

    const float step = 0.000001f;

    for (int i = 0; i < x_size; i++)
    {
        x[i] = i * step;
    }

    const int block_dim = 256;
    const int grid_dim = (x_size + block_dim - 1) / block_dim;

    sine<<<grid_dim, block_dim>>>(x_size, x);
    CUDA_CHECK(hipPeekAtLastError());

    CUDA_CHECK(hipDeviceSynchronize());

    float max_error = 0.0f;

    for (int i = 0; i < x_size; i++)
    {
        const float expected = sin(i * step);

        max_error = fmax(max_error, fabs(x[i] - expected));
    }

    printf("Max error: %.10f\n", max_error);

    CUDA_CHECK(hipFree(x));

    return 0;
}
