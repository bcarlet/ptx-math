#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <cmath>
#include <cstring>
#include <limits>

using float_limits = std::numeric_limits<float>;

// sanity checks
static_assert(CHAR_BIT == 8, "CHAR_BIT != 8");
static_assert(sizeof(float) == 4, "sizeof(float) != 4");
static_assert(!float_limits::traps, "float generates traps");

#include "util/stopwatch.hpp"
#include "util/running_stats.hpp"
#include "cuda_util.hpp"
#include "ptx.hpp"

static constexpr uint32_t BATCH_SIZE = UINT32_C(1) << 20;
static constexpr uint32_t BATCH_COUNT = (UINT64_C(1) << 32) / BATCH_SIZE;

static constexpr int BLOCK_DIM = 1 << 8;
static constexpr int GRID_DIM = BATCH_SIZE / BLOCK_DIM;

template<ptx_instruction I>
__global__
static void map(int n, float *x)
{
    GRID_STRIDE_LOOP(i, n)
    {
        ptx_asm<I>::exec(x + i);
    }
}

static void initialize_batch(uint32_t batch, float *x)
{
    uint32_t val = batch * BATCH_SIZE;

    for (uint32_t i = 0; i < BATCH_SIZE; i++)
    {
        memcpy(x + i, &val, 4u);
        val++;
    }
}

struct comp_stats
{
    void accumulate(float a, float b)
    {
        if (isfinite(a) && isfinite(b))
            error.accumulate(fabs((double)b - a));

        if (memcmp(&a, &b, sizeof(float)) == 0)
            num_exact++;
    }

    running_stats<double> error;
    unsigned long long num_exact = 0u;
};

static void compare_batch(uint32_t batch, const float *x, float (*f)(float), comp_stats &stats)
{
    uint32_t val = batch * BATCH_SIZE;

    for (uint32_t i = 0; i < BATCH_SIZE; i++)
    {
        float fval;
        memcpy(&fval, &val, 4u);

        stats.accumulate(x[i], f(fval));

        val++;
    }
}

int main()
{
    float *x;
    CUDA_CHECK(hipMallocManaged(&x, BATCH_SIZE * sizeof(float)));

    stopwatch<double, std::milli> timer;
    running_stats<double> time;
    comp_stats stats;

    for (uint32_t batch = 0; batch < BATCH_COUNT; batch++)
    {
        if (batch % (BATCH_COUNT / 8) == 0)
            printf("On batch: %u\n", batch);

        initialize_batch(batch, x);

        timer.reset();

        map<ptx_instruction::SIN_APPROX_F32><<<GRID_DIM, BLOCK_DIM>>>(BATCH_SIZE, x);
        CUDA_CHECK(hipPeekAtLastError());

        CUDA_CHECK(hipDeviceSynchronize());

        time.accumulate(timer.elapsed());
        compare_batch(batch, x, sinf, stats);
    }

    printf("GPU batch time (ms): min=%f, max=%f, avg=%f\n", time.min, time.max, time.average());
    printf("Finite error: max=%.15f, avg=%.15f\n", stats.error.max, stats.error.average());
    printf("Bit-exact: %llu\n", stats.num_exact);

    CUDA_CHECK(hipFree(x));

    return 0;
}
