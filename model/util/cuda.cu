#include "cuda.hpp"

#include <cstdlib>
#include <cstdio>

void print_devices()
{
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));

    hipDeviceProp_t props;

    for (int device = 0; device < device_count; device++)
    {
        CUDA_CHECK(hipGetDeviceProperties(&props, device));

        fprintf(stdout, "Device %d: %s (sm_%d%d)\n",
                device, props.name, props.major, props.minor);
    }
}

void cuda_error_check(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA error in file %s on line %d: %s\n",
                file, line, hipGetErrorString(code));

        exit(-1);
    }
}
