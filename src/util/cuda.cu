#include "cuda.cuh"

#include <cstdlib>
#include <iostream>

void cuda_check(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "CUDA error in file " << file << " on line " << line
                  << ": " << hipGetErrorString(code) << '\n';

        std::exit(EXIT_FAILURE);
    }
}
