#include "cuda.cuh"

#include <cstdlib>
#include <iostream>

void cuda_error_check(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "CUDA error in file " << file << " on line " << line
                  << ": " << hipGetErrorString(code) << '\n';

        std::exit(-1);
    }
}
