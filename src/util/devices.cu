#include "devices.hpp"
#include "cuda.hpp"

std::vector<hipDeviceProp_t> get_device_props()
{
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));

    std::vector<hipDeviceProp_t> props(device_count);

    for (int device = 0; device < device_count; device++)
    {
        CUDA_CHECK(hipGetDeviceProperties(props.data() + device, device));
    }

    return props;
}

std::ostream &operator<<(std::ostream &stream, const hipDeviceProp_t &prop)
{
    stream << prop.name << " (sm_" << prop.major << prop.minor << ')';

    return stream;
}
